#include "hip/hip_runtime.h"
#include "stdio.h"
#include "string.h"
/////////////////////////////////////
//Database Compare TC matrix Calculation
/////////////////////////////////////
int* d_TargetMagArray;
int* d_TargetMatrix;
int* d_RefMatrix;
float* d_TCMatrix;
__constant__ int constMagArray[129];
int* d_MaxTCIndex;
float* d_MaxTCValue;

#define SUCCESS 1
#define UNKNOWN_ERROR 0
#define MEMORY_ALLOC_FAIL 2

extern "C" int GetDeviceInfo(char* pDeviceName, size_t* pTotalMem, int* pClockRate, int* pNumProcessor)
{
	int nDevice;
	if(hipGetDeviceCount(&nDevice)!=hipSuccess)
		nDevice=-1;
	if(nDevice>0)
	{
		hipDeviceProp_t myProp;
		hipSetDevice(0);
		hipGetDeviceProperties(&myProp,0);
		strcpy(pDeviceName,myProp.name);
		*pTotalMem=myProp.totalGlobalMem;
		*pClockRate=myProp.clockRate;
		*pNumProcessor=myProp.multiProcessorCount;
	}

	return nDevice;
}

__global__ void TCMatrixKernel(int nTargetCount, int nTargetRealCount, int nTargetDim, int* d_TargetMagArray, int* d_TargetMatrix,
						int nRefCount, int nRefDim, int* d_RefMatrix,float* d_TCMatrix,int Offset,int TCMatrixPitch)
{
	if(blockIdx.x+Offset<nRefCount)
	{
		__shared__ int sharedQuery[512];
		int RefMag=constMagArray[blockIdx.x];
		int TargetMag;
		if(threadIdx.x<RefMag)
		{
			sharedQuery[threadIdx.x]=d_RefMatrix[(blockIdx.x+Offset)*nRefDim + threadIdx.x];
		}
		__syncthreads();
		int TargetIndex,RefIndex,UnionCount;
		int buffer;
		int index;
		
		for(int i=0;i<(nTargetCount+511)/512;i++)
		{
			index=threadIdx.x+i*512;
			if(index<nTargetRealCount)
			{
				TargetMag=d_TargetMagArray[index];
				RefIndex=UnionCount=0;
				for(TargetIndex=0;TargetIndex<TargetMag ;TargetIndex++)
				{
					buffer=d_TargetMatrix[TargetIndex*nTargetCount+index];
					while(sharedQuery[RefIndex]<buffer && RefIndex<RefMag)
						RefIndex++;
					if(buffer==sharedQuery[RefIndex])
						UnionCount++;
				}
				d_TCMatrix[blockIdx.x*TCMatrixPitch+index]=(float)UnionCount/(RefMag+TargetMag-UnionCount);
			}
		}
	}
	
}
__global__ void ExcludeSelf(int nTargetRealCount, float* d_TCMatrix, int Offset, int TCMatrixPitch)
{
	if(threadIdx.x<128)
	{
		int index=Offset+threadIdx.x;
		if(index<nTargetRealCount)
		{
			d_TCMatrix[threadIdx.x*TCMatrixPitch+index]=0;
		}
	}
}
__global__ void GetMaxTCKernel(int nTargetRealCount, float* d_MaxTCValue, int* d_MaxTCIndex, float* d_TCMatrix, int TCMatrixPitch)
{
	__shared__ int sMaxTCIndex[256];
	__shared__ float sMaxTCValue[256];
	float buffer;
	
	unsigned int tid=threadIdx.x;
	if(tid<nTargetRealCount)
	{
		sMaxTCValue[tid]=d_TCMatrix[blockIdx.x*TCMatrixPitch+tid];
		sMaxTCIndex[tid]=tid;
	}
	tid+=blockDim.x;
	while(tid<nTargetRealCount)
	{
		buffer=d_TCMatrix[blockIdx.x*TCMatrixPitch+tid];
		if(buffer>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=buffer;
			sMaxTCIndex[threadIdx.x]=tid;
		}
		tid+=blockDim.x;
	}
	__syncthreads();
	if(threadIdx.x<128)
	{
		if(sMaxTCValue[threadIdx.x+128]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+128];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+128];

		}
		__syncthreads();
	}
	if(threadIdx.x<64)
	{
		if(sMaxTCValue[threadIdx.x+64]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+64];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+64];

		}
		__syncthreads();
	}
	if(threadIdx.x<32)
	{
		if(sMaxTCValue[threadIdx.x+32]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+32];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+32];

		}
	}
	if(threadIdx.x<16)
	{
		if(sMaxTCValue[threadIdx.x+16]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+16];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+16];

		}
	}
	if(threadIdx.x<8)
	{
		if(sMaxTCValue[threadIdx.x+8]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+8];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+8];

		}
	}
	if(threadIdx.x<4)
	{
		if(sMaxTCValue[threadIdx.x+4]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+4];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+4];

		}
	}
	if(threadIdx.x<2)
	{
		if(sMaxTCValue[threadIdx.x+2]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+2];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+2];

		}
	}
	if(threadIdx.x<1)
	{
		if(sMaxTCValue[threadIdx.x+1]>sMaxTCValue[threadIdx.x])
		{
			sMaxTCValue[threadIdx.x]=sMaxTCValue[threadIdx.x+1];
			sMaxTCIndex[threadIdx.x]=sMaxTCIndex[threadIdx.x+1];

		}
		d_MaxTCValue[blockIdx.x]=sMaxTCValue[threadIdx.x];
		d_MaxTCIndex[blockIdx.x]=sMaxTCIndex[threadIdx.x];
	}
	

}
/////////////////////////////////////////////////////
// Calculate TC Histogram
////////////////////////////////////////////////////

__global__ void ComputeTCHist(float* d_Data, int nRefCount, int* d_TCHist,int batch)
{
	__shared__ unsigned int s_Hist[3840];
	int WarpId=threadIdx.x>>5;
	int ThreadIndex=threadIdx.x-WarpId*32;
	int bin;
	int i,j,index;
	for(j=0;j<5;j++)
	{
		for(i=0;i<20;i++)
			s_Hist[i*192+threadIdx.x]=0;
		__syncthreads();
		for(i=0;i<batch;i++)
		{
			index=threadIdx.x+192*i+blockIdx.x*batch*192;
			if(index<nRefCount)
			{
				bin=floorf(d_Data[index]*99.0001f);
				
				if(bin>=j*20 && bin<j*20+20)
				{
					bin=bin-j*20;
					s_Hist[WarpId*640+bin*32+ThreadIndex]++;
				}
				
			}
		}
		__syncthreads();
		for(i=1;i<6;i++)
		{
			s_Hist[threadIdx.x]+=s_Hist[threadIdx.x+i*640];
			s_Hist[threadIdx.x+192]+=s_Hist[threadIdx.x+i*640+192];
			s_Hist[threadIdx.x+384]+=s_Hist[threadIdx.x+i*640+384];
			if(threadIdx.x<64)
				s_Hist[threadIdx.x+576]+=s_Hist[threadIdx.x+i*640+576];
		}
		__syncthreads();
		if(threadIdx.x<20)
		{
			for(i=1;i<32;i++)
			{
				//bank conflict
				s_Hist[threadIdx.x*32]+=s_Hist[threadIdx.x*32+i];
			}
			d_TCHist[blockIdx.x*128+threadIdx.x+j*20]=s_Hist[threadIdx.x*32];
			
		}
		__syncthreads();
	}
}
__global__ void addByte(int* d_Data,int dim,int N)
{
	int i;
	if(threadIdx.x<N)
	{
		for(i=1;i<dim;i++)
		{
			d_Data[threadIdx.x]+=d_Data[threadIdx.x+N*i];
		}
	}
}
extern "C" int SPADBTC(int nTargetCount, int nTargetRealCount, int nTargetDim, int* pTargetMagArray, int* pTargetMatrix,
						int nRefCount, int nRefRealCount, int nRefDim, int* pRefMagArray, int* pRefMatrix, int SelfCompare, int* res,int* pProgress)
{
	
	int TCMatrixPitch=(nTargetCount+15)/16*16;
	float* d_MaxTCArray;
	int* d_MaxTCIndexArray;
	hipError_t msg;
	msg=hipMalloc((void**)&d_TargetMagArray,sizeof(int)*nTargetCount);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL;
	msg=hipMalloc((void**)&d_TargetMatrix,sizeof(int)*nTargetCount*nTargetDim);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+1;
	msg=hipMalloc((void**)&d_RefMatrix,sizeof(int)*nRefCount*nRefDim);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+2;
	msg=hipMalloc((void**)&d_TCMatrix,sizeof(float)*TCMatrixPitch*128);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+3;
	msg=hipMalloc((void**)&d_MaxTCIndexArray,sizeof(float)*(nRefCount+127)/128*128);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+4;
	msg=hipMalloc((void**)&d_MaxTCArray,sizeof(float)*(nRefCount+127)/128*128);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+5;

	hipMemcpy(d_TargetMagArray, pTargetMagArray, sizeof(int)*nTargetCount, hipMemcpyHostToDevice);
	hipMemcpy(d_TargetMatrix, pTargetMatrix, sizeof(int)*nTargetCount*nTargetDim, hipMemcpyHostToDevice);
	hipMemcpy(d_RefMatrix, pRefMatrix, sizeof(int)*nRefCount*nRefDim, hipMemcpyHostToDevice);
	int threadsPerBlock = 512;
    int blocksPerGrid = 128;
	int Offset;int i;
	

	for(i=0;i<(nRefCount+127)/128;i++)
	{
		*pProgress=i+1;
		Offset=i*128;
		if(Offset+128<nRefCount)
			msg=hipMemcpyToSymbol(HIP_SYMBOL(constMagArray),pRefMagArray+Offset,sizeof(int)*128,0,hipMemcpyHostToDevice);
		else
			msg=hipMemcpyToSymbol(HIP_SYMBOL(constMagArray),pRefMagArray+Offset,sizeof(int)*(nRefCount-Offset),0,hipMemcpyHostToDevice);
		if(msg!=hipSuccess)
			return UNKNOWN_ERROR;
		TCMatrixKernel<<<blocksPerGrid, threadsPerBlock>>>(nTargetCount, nTargetRealCount, nTargetDim, d_TargetMagArray, d_TargetMatrix,
							nRefCount, nRefDim, d_RefMatrix, d_TCMatrix, Offset,TCMatrixPitch);
		hipDeviceSynchronize();
		if(SelfCompare)
		{
			ExcludeSelf<<<1,192>>>(nTargetRealCount, d_TCMatrix, Offset, TCMatrixPitch);
			hipDeviceSynchronize();
		}
		GetMaxTCKernel<<<128,256>>>(nTargetRealCount, d_MaxTCArray+Offset, d_MaxTCIndexArray+Offset, d_TCMatrix, TCMatrixPitch);
	}
	
	//Histogram Calculation
	hipDeviceSynchronize();
	int* d_TCHist;
	msg=hipMalloc((void**)&d_TCHist,sizeof(int)*128*16);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+6;
	
	ComputeTCHist<<<16,192>>>(d_MaxTCArray,nRefCount,d_TCHist,(nRefCount+3071)/3072);
	hipDeviceSynchronize();

	addByte<<<1,192>>>(d_TCHist,16,128);
	hipDeviceSynchronize();
	hipMemcpy(res, d_TCHist, sizeof(int)*100, hipMemcpyDeviceToHost);

	hipFree(d_TCHist);
	hipFree(d_TCMatrix);
	hipFree(d_TargetMagArray);
	hipFree(d_TargetMatrix);
	hipFree(d_RefMatrix);
	hipFree(d_MaxTCArray);
	hipFree(d_MaxTCIndexArray);

	return SUCCESS;

}

///////////////////////////////////////////////////////
//TC calculated based on integer fingerprint and lookup table
/////////////////////////////////////////////////////////
__global__ void IntTCMatrixKernel(int nTargetCount, int nTargetRealCount, int nTargetDim, int* d_TargetMagArray, int* d_TargetMatrix,
						int nRefCount, int nRefDim, int* d_RefMatrix,float* d_TCMatrix,int Offset,int TCMatrixPitch /*,int* d_LookupTable*/)
{
	if(blockIdx.x+Offset<nRefCount)
	{
		extern __shared__ int sharedQuery[];
		int RefMag=constMagArray[blockIdx.x];
		int TargetMag;
		int index,i,j,UnionCount,buffer;
		if(threadIdx.x<nRefDim)
		{
			sharedQuery[threadIdx.x]=d_RefMatrix[(blockIdx.x+Offset)*nRefDim + threadIdx.x];
		}
		__syncthreads();
		
		for(i=0;i<(nTargetCount+255)/256;i++)
		{
			index=threadIdx.x+i*256;
			if(index<nTargetCount)
			{
				UnionCount=0;
				TargetMag=d_TargetMagArray[index];
				for(j=0;j<nTargetDim;j++)
				{
					buffer=d_TargetMatrix[j*nTargetCount+index]&sharedQuery[j];
					UnionCount+=__popc(buffer);
				}
				d_TCMatrix[blockIdx.x*TCMatrixPitch+index]=(float)UnionCount/(RefMag+TargetMag-UnionCount);
			}		
		}
	}
}

extern "C" int INTDBTC(int nTargetCount, int nTargetRealCount, int nTargetDim, int* pTargetMagArray, int* pTargetMatrix,
						int nRefCount, int nRefRealCount, int nRefDim, int* pRefMagArray, int* pRefMatrix, int SelfCompare, int* res, int* pProgress)
{
	float* d_MaxTCArray;
	int* d_MaxTCIndexArray;
	int TCMatrixPitch=(nTargetCount+15)/16*16;

	hipError_t msg;
	msg=hipMalloc((void**)&d_TargetMagArray,sizeof(int)*nTargetCount);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL;
	msg=hipMalloc((void**)&d_TargetMatrix,sizeof(int)*nTargetCount*nTargetDim);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+1;
	msg=hipMalloc((void**)&d_RefMatrix,sizeof(int)*nRefCount*nRefDim);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+2;
	msg=hipMalloc((void**)&d_TCMatrix,sizeof(float)*TCMatrixPitch*128);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+3;

	msg=hipMalloc((void**)&d_MaxTCIndexArray,sizeof(float)*(nRefCount+127)/128*128);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+4;

	msg=hipMalloc((void**)&d_MaxTCArray,sizeof(float)*(nRefCount+127)/128*128);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+5;
	

	hipMemcpy(d_TargetMagArray, pTargetMagArray, sizeof(int)*nTargetCount, hipMemcpyHostToDevice);
	hipMemcpy(d_TargetMatrix, pTargetMatrix, sizeof(int)*nTargetCount*nTargetDim, hipMemcpyHostToDevice);
	hipMemcpy(d_RefMatrix, pRefMatrix, sizeof(int)*nRefCount*nRefDim, hipMemcpyHostToDevice);
	int threadsPerBlock = 256;
    int blocksPerGrid = 128;
	int Offset;int i;
	
	
	for(i=0;i<(nRefCount+127)/128;i++)
	{
		*pProgress=i+1;
		Offset=i*128;
		if(Offset+128<nRefCount)
			msg=hipMemcpyToSymbol(HIP_SYMBOL(constMagArray),pRefMagArray+Offset,sizeof(int)*128,0,hipMemcpyHostToDevice);
		else
			msg=hipMemcpyToSymbol(HIP_SYMBOL(constMagArray),pRefMagArray+Offset,sizeof(int)*(nRefCount-Offset),0,hipMemcpyHostToDevice);
		if(msg!=hipSuccess)
			return UNKNOWN_ERROR;
		IntTCMatrixKernel<<<blocksPerGrid, threadsPerBlock,nRefDim*sizeof(int)>>>(nTargetCount, nTargetRealCount, nTargetDim, d_TargetMagArray, d_TargetMatrix,
							nRefCount, nRefDim, d_RefMatrix, d_TCMatrix, Offset,TCMatrixPitch);
		hipDeviceSynchronize();
		if(SelfCompare)
		{
			ExcludeSelf<<<1,192>>>(nTargetRealCount, d_TCMatrix, Offset, TCMatrixPitch);
			hipDeviceSynchronize();
		}
		GetMaxTCKernel<<<128,256>>>(nTargetRealCount, d_MaxTCArray+Offset, d_MaxTCIndexArray+Offset, d_TCMatrix, TCMatrixPitch);
		
	}
	hipDeviceSynchronize();
	int* d_TCHist;
	msg=hipMalloc((void**)&d_TCHist,sizeof(int)*128*16);
	if(msg!=hipSuccess)
		return MEMORY_ALLOC_FAIL+7;
	

	
	ComputeTCHist<<<16,192>>>(d_MaxTCArray,nRefCount,d_TCHist,(nRefCount+3071)/3072);
	hipDeviceSynchronize();

	addByte<<<1,192>>>(d_TCHist,16,128);
	hipDeviceSynchronize();
	hipMemcpy(res, d_TCHist, sizeof(int)*100, hipMemcpyDeviceToHost);


	hipFree(d_TCHist);
	hipFree(d_TCMatrix);
	hipFree(d_TargetMagArray);
	hipFree(d_TargetMatrix);
	hipFree(d_RefMatrix);
	hipFree(d_MaxTCArray);
	hipFree(d_MaxTCIndexArray);

	return SUCCESS;

}
